// This is not my code, this comes from NVIDIA's "CUDA C++ Programming Guide"
/* 
This code allocates enough memory for a 2 dimensional array with integers for the device. To optimize 
the code, reference this optimized code and allocate memory using hipMallocPitch to create a 2 
dimensional array on the device.
*/
int width = 64, height = 64; // these two variable declare the dimensions of the 2D array for the device
float* devPtr; // this variable is a pointer that will point to the first value of the allocated 2D array on the device
size_t pitch; // this variable will keep track of the actual length of each row of the array in the memory
hipMallocPitch(&devPtr, &pitch,
                width * sizeof(float), height);
