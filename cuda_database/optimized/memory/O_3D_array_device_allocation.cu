// This is not my code, this comes from NVIDIA's "CUDA C++ Programming Guide"
/* 
This code allocates enough memory for a 3 dimensional array with integers for the device. To optimize 
the code, reference this optimized code and allocate memory using hipMalloc3D to create a 3 
dimensional array on the device.
*/
int width = 64, height = 64, depth = 64; // these variables declare the dimensions of the 3D array
hipExtent extent = make_hipExtent(width * sizeof(float), height, depth); // these lines create a hipExtent structure called 'extent' that uses make_hipExtent to conveniently create an extent that defines 3 dimensions in memory space, it takes the three dimension variables as input
hipPitchedPtr devPitchedPtr; 
hipMalloc3D(&devPitchedPtr, extent);
