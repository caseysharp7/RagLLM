// This is not my code, this comes from NVIDIA's "CUDA C++ Programming Guide"

for (int i = 0; i < 2; ++i)
    hipMemcpyAsync(inputDevPtr + i * size, hostPtr + i * size,
                    size, hipMemcpyHostToDevice, stream[i]);
for (int i = 0; i < 2; ++i)
    MyKernel<<<100, 512, 0, stream[i]>>>
          (outputDevPtr + i * size, inputDevPtr + i * size, size);
for (int i = 0; i < 2; ++i)
    hipMemcpyAsync(hostPtr + i * size, outputDevPtr + i * size,
                    size, hipMemcpyDeviceToHost, stream[i]);
