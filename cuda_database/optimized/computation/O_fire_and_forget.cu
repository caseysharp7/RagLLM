#include "hip/hip_runtime.h"
// This is not my code, this comes from NVIDIA's "CUDA C++ Programming Guide"

__global__ void launchFireAndForgetGraph(hipGraphExec_t graph) {
    hipGraphLaunch(graph, cudaStreamGraphFireAndForget);
}

void graphSetup() {
    hipGraphExec_t gExec1, gExec2;
    hipGraph_t g1, g2;

    // Create, instantiate, and upload the device graph.
    create_graph(&g2);
    hipGraphInstantiate(&gExec2, g2, hipGraphInstantiateFlagDeviceLaunch);
    hipGraphUpload(gExec2, stream);

    // Create and instantiate the launching graph.
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    launchFireAndForgetGraph<<<1, 1, 0, stream>>>(gExec2);
    hipStreamEndCapture(stream, &g1);
    hipGraphInstantiate(&gExec1, g1);

    // Launch the host graph, which will in turn launch the device graph.
    hipGraphLaunch(gExec1, stream);
}
