// This is not my code, this comes from NVIDIA's "CUDA C++ Programming Guide"

hipGraphExec_t graphExec = NULL;

for (int i = 0; i < 10; i++) {
    hipGraph_t graph;
    hipGraphExecUpdateResult updateResult;
    hipGraphNode_t errorNode;

    // In this example we use stream capture to create the graph.
    // You can also use the Graph API to produce a graph.
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

    // Call a user-defined, stream based workload, for example
    do_cuda_work(stream);

    hipStreamEndCapture(stream, &graph);

    // If we've already instantiated the graph, try to update it directly
    // and avoid the instantiation overhead
    if (graphExec != NULL) {
        // If the graph fails to update, errorNode will be set to the
        // node causing the failure and updateResult will be set to a
        // reason code.
        hipGraphExecUpdate(graphExec, graph, &errorNode, &updateResult);
    }

    // Instantiate during the first iteration or whenever the update
    // fails for any reason
    if (graphExec == NULL || updateResult != hipGraphExecUpdateSuccess) {

        // If a previous update failed, destroy the hipGraphExec_t
        // before re-instantiating it
        if (graphExec != NULL) {
            hipGraphExecDestroy(graphExec);
        }
        // Instantiate graphExec from graph. The error node and
        // error message parameters are unused here.
        hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
    }

    hipGraphDestroy(graph);
    hipGraphLaunch(graphExec, stream);
    hipStreamSynchronize(stream);
}
