// This is my own code, though it is closely related to the optimized example

for(int i = 0; i < 2; ++i){
    hipMemcpy(inputDevPtr + i * size, hostPtr + i * size, size, hipMemcpyHostToDevice);
}
for(int i = 0; i < 2; ++i){
    kernel<<<100, 512>>>(outputDevPtr + i * size, inputDevPtr + i * size, size);
}
for(int i = 0; i < 2; ++i){
    hipMemcpy(hostPtr + i * size, outputDevPtr + i * size, size, hipMemcpyDeviceToHost);
}
