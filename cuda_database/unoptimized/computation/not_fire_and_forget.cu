void graph(){
    hipGraph_t graph;
    hipGraphExec_t graph_exec;

    create_graph(&graph);
    hipGraphInstantiate(&graph_exec, graph);
    hipGraphUpload(graph_exec, stream);

    hipGraphLaunch(graph_exec, stream);
}
