// My justification for this unoptimized version is that a confused user might try to use multiple streams to create a parallel graphlike structure without the understanding 
// that the synchronization damages the ability of the streams to launch kernels in parallel

hipStream_t stream1, stream2, stream3;
hipStreamCreate(&stream1);
hipStreamCreate(&stream2);
hipStreamCreate(&stream3);

for (int i = 0; i < num; i++) {
    cuda_function(stream1);
    cuda_function(stream2);
    cuda_function(stream3);

    hipStreamSynchronize(stream);
}
