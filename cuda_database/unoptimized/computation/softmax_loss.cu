#include "hip/hip_runtime.h"
__global__ void
softmax_loss_kernel(float *reduced_loss, float *predict, float *target, float *workspace, int num_outputs)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    float loss = 0.0;

    for(int j = 0; j < num_outputs; j++)
        loss += target[batch_idx * num_outputs + j] * logf(predict[batch_idx * num_outputs + j]);
    workspace[batch_idx] = -loss;

    float data_loss = 0;
    for(int stride = blockDim.x; stride > 0; stride--){
        if(threadIdx.x == 0){
            data_loss += workspace[stride];
        }
    }

    if (threadIdx.x == 0) {
        reduced_loss[blockIdx.x] = data_loss;
    }
}
