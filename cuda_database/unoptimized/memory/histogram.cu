
#include <hip/hip_runtime.h>
__global__ void func1(int *var1, int *var2, int var3)
{
    int var4 = blockDim.x * blockIdx.x + threadIdx.x;

    if(var3 > var4){
        int var5 = var1[var4];

        atomicAdd(&var2[var5], 1);
    }
}