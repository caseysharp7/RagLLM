
#include <hip/hip_runtime.h>
__global__ void histogram(int *input_data, int *bins, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(size > i){
        int memory = input_data[i];

        atomicAdd(&bins[memory], 1);
    }
}
